#include "hip/hip_runtime.h"
﻿#include "file_system.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

__device__ __managed__ u32 gtime = 0;

/*
  Description:   File system initialization
  Input:  Args
  Output:   N/A
*/
__device__ void fs_init(FileSystem *fs, uchar *volume, int SUPERBLOCK_SIZE,
							int FCB_SIZE, int FCB_ENTRIES, int VOLUME_SIZE,
							int STORAGE_BLOCK_SIZE, int MAX_FILENAME_SIZE, 
							int MAX_FILE_NUM, int MAX_FILE_SIZE, int FILE_BASE_ADDRESS)
{
  // init variables
  fs->volume = volume;

  // init constants (Unit: byte / B)
  fs->SUPERBLOCK_SIZE = SUPERBLOCK_SIZE;  // 4096
  fs->FCB_SIZE = FCB_SIZE;  //  32
  fs->FCB_ENTRIES = FCB_ENTRIES;    // 1024
  fs->STORAGE_SIZE = VOLUME_SIZE;   // 1085440
  fs->STORAGE_BLOCK_SIZE = STORAGE_BLOCK_SIZE;    //32
  fs->MAX_FILENAME_SIZE = MAX_FILENAME_SIZE;    // 20
  fs->MAX_FILE_NUM = MAX_FILE_NUM;    // 1024
  fs->MAX_FILE_SIZE = MAX_FILE_SIZE;    // 1048576
  fs->FILE_BASE_ADDRESS = FILE_BASE_ADDRESS;    // 36864

  // Extra intermediate variable space (8 bytes)
  fs->MODIFY_TIME = 0;
  fs->CREATE_TIME = 0;
  fs->VALID_BLOCK = 0;
}

/*
  Description:   Copy content from source to target
  Input:    uchar pointer (pointing to the source and target)
  Output:   N/A 
*/
__device__ void memcpy(uchar *target, uchar *source, int size){
  for(int i = 0; i < size; i++){
    target[i] = source[i];
  }
}

/*
  Description:   Compare the source and target with equal size (used in comparing file names)
  Input:  uchar pointer (pointing to the source and target)
  Output:   1 -> different; 0-> identical
*/
__device__ u32 memcmp(uchar *target, uchar *source, int size){
  for(int i = 0; i < size; i++){
    if(target[i] != source[i]) return 1;
    else if(target[i] == '\0') return 0;
  }
  return 0;
}

/*
  Description:   Read the # FCB's valid bit. 
  Input:    FCB_address
  Output:   Valid = 1, Invalid = 0.
*/
__device__ u32 FCB_read_validbit(FileSystem *fs, u32 FCB_address){
  uchar *target = &fs->volume[fs->SUPERBLOCK_SIZE + FCB_address*fs->FCB_SIZE + 25];
  return (*target >> 5) & 0b00000001;
}

/*
  Description:   Set the FCB permission bit. Valid = 1, Invalid = 0.
  Input:  Valid = 1, Invalid = 0, FCB_address
  Output:   N/A
*/
__device__ void FCB_set_validbit(FileSystem *fs, u32 FCB_address, u32 value){
  uchar *target = &fs->volume[fs->SUPERBLOCK_SIZE + FCB_address*fs->FCB_SIZE + 25];
  fs->volume[fs->SUPERBLOCK_SIZE + FCB_address*fs->FCB_SIZE + 25] = (*target & 0b11011111) + (value << 5);
}

/*
  Description:   Read the FCB filename
  Input:  output pointer, FCB_address
  Output:   N/A
*/
__device__ void FCB_read_filename(FileSystem *fs, u32 FCB_address, uchar *output){
  uchar *source = &fs->volume[fs->SUPERBLOCK_SIZE + FCB_address*fs->FCB_SIZE];
  memcpy(output, source, 20);
}

/*
  Description:   Set the FCB filename
  Input:  input pointer, FCB_address
  Output:   N/A
*/
__device__ void FCB_set_filename(FileSystem *fs, u32 FCB_address, uchar *input){
  uchar *target = &fs->volume[fs->SUPERBLOCK_SIZE + FCB_address*fs->FCB_SIZE];
  memcpy(target, input, 20);
}

/*
  Description:   Read the FCB starting point address (Unit: block)
  Input:  FCB_address
  Output:   start block number
*/
__device__ u32 FCB_read_start(FileSystem *fs, u32 FCB_address){
  uchar *source = &fs->volume[fs->SUPERBLOCK_SIZE + FCB_address*fs->FCB_SIZE+20];
  u32 result;
  memcpy((uchar*)&result, source, 2);
  return result;
}

/*
  Description:   Set the FCB starting point address (Unit: block)
  Input:  start block number, FCB_address
  Output:   N/A
*/
__device__ void FCB_set_start(FileSystem *fs, u32 FCB_address, u32 value){
  uchar *target = &fs->volume[fs->SUPERBLOCK_SIZE + FCB_address*fs->FCB_SIZE+20];
  memcpy(target, (uchar*)&value, 2);
}

/*
  Description:   Read the FCB size (Unit: bytes)
  Input:  FCB_address
  Output:   size
*/
__device__ u32 FCB_read_size(FileSystem *fs, u32 FCB_address){
  uchar *source = &fs->volume[fs->SUPERBLOCK_SIZE + FCB_address*fs->FCB_SIZE+22];
  u32 result;
  memcpy((uchar*)&result, source, 3);
  return result;
}

/*
  Description:   Set the FCB size (Unit: bytes)
  Input:  size, FCB_address
  Output:   N/A
*/
__device__ void FCB_set_size(FileSystem *fs, u32 FCB_address, u32 value){
  uchar *target = &fs->volume[fs->SUPERBLOCK_SIZE + FCB_address*fs->FCB_SIZE+22];
  memcpy(target, (uchar*)&value, 3);
}

/*
  Description:   Read the FCB Last modified time
  Input: FCB_address
  Output: Last modified time
*/
__device__ u32 FCB_read_ltime(FileSystem *fs, u32 FCB_address){
  uchar *source = &fs->volume[fs->SUPERBLOCK_SIZE + FCB_address*fs->FCB_SIZE+26];
  u32 result;
  memcpy((uchar*)&result, source, 3);
  return result;
}

/*
  Description:   Compact the last modified time series
  Input:  N/A
  Output:   N/A
*/
__device__ void ltime_compact(FileSystem *fs){
  u32 current_min_ltime = pow(2,25);
  u32 current_address = 0;
  u32 last_target = 0;
  u32 current_ltime = 0;
  for(u32 i = 0; i < fs->VALID_BLOCK; i++){
    current_min_ltime = pow(2,25);
    for(int FCB_address = 0; FCB_address < fs->FCB_ENTRIES; FCB_address++){
      if(!FCB_read_validbit(fs, FCB_address)) continue;
      current_ltime = FCB_read_ltime(fs, FCB_address);
      
      if( current_ltime > last_target && current_ltime < current_min_ltime ){
        current_address = FCB_address;
        current_min_ltime = current_ltime;
      }
    }
    last_target++;
    uchar *target = &fs->volume[fs->SUPERBLOCK_SIZE + current_address*fs->FCB_SIZE+26];
    memcpy(target, (uchar*)&last_target, 3);
  }
  fs->MODIFY_TIME = last_target + 1;
}

/*
  Description:   Set the FCB Last modified time
  Input:  FCB_address, Last modified time
  Output: N/A
*/
__device__ void FCB_set_ltime(FileSystem *fs, u32 FCB_address){
  if(fs->MODIFY_TIME > pow(2,24) - 30) ltime_compact(fs); 
  uchar *target = &fs->volume[fs->SUPERBLOCK_SIZE + FCB_address*fs->FCB_SIZE+26];
  memcpy(target, (uchar*)&fs->MODIFY_TIME, 3);
  fs->MODIFY_TIME++;
}

/*
  Description:   Read the FCB created time
  Input:  FCB_address
  Output:   Create time
*/
__device__ u32 FCB_read_ctime(FileSystem *fs, u32 FCB_address){
  uchar *source = &fs->volume[fs->SUPERBLOCK_SIZE + FCB_address*fs->FCB_SIZE+29];
  u32 result;
  memcpy((uchar*)&result, source, 3);
  return result;
}

/*
  Description:   Compact the create time series
  Input:  N/A
  Output:   N/A
*/
__device__ void ctime_compact(FileSystem *fs){
  u32 current_min_ctime = pow(2,25);
  u32 current_address = 0;
  u32 last_target = 0;
  u32 current_ctime = 0;
  for(u32 i = 0; i < fs->VALID_BLOCK; i++){
    current_min_ctime = pow(2,25);
    for(int FCB_address = 0; FCB_address < fs->FCB_ENTRIES; FCB_address++){
      if(!FCB_read_validbit(fs, FCB_address)) continue;
      current_ctime = FCB_read_ctime(fs, FCB_address);
      if( current_ctime > last_target && current_ctime < current_min_ctime ){
        current_address = FCB_address;
        current_min_ctime = current_ctime;
      }
    }
    last_target++;
    uchar *target = &fs->volume[fs->SUPERBLOCK_SIZE + current_address*fs->FCB_SIZE+29];
    memcpy(target, (uchar*)&last_target, 3);
  }
  fs->MODIFY_TIME = last_target + 1;
}

/*
  Description:   Set the FCB created time
  Input:  FCB_address
  Output:   N/A
*/
__device__ void FCB_set_ctime(FileSystem *fs, u32 FCB_address){
  if(fs->MODIFY_TIME > pow(2,24) - 30) ctime_compact(fs); 
  uchar *target = &fs->volume[fs->SUPERBLOCK_SIZE + FCB_address*fs->FCB_SIZE+29];
  memcpy(target, (uchar*)&fs->CREATE_TIME, 3);
  fs->CREATE_TIME++;
}

/*
  Description:   For testing only, printing out all FCB info.
  Input:    N/A
  Output:   N/A
*/

__device__ void print_FCB(FileSystem *fs){
  u32 FCB_address;
  uchar file_name[20];
  printf("===============PRINTING_FCB_BLOCK_INFO=====================\n");
  printf("Valid blocks = %d\n",fs->VALID_BLOCK);
  for(FCB_address = 0; FCB_address < fs->FCB_ENTRIES; FCB_address++){
    if(!FCB_read_validbit(fs,FCB_address)) continue;
    FCB_read_filename(fs, FCB_address, file_name);
    printf("Block %5d, name = %20s,start = %10d, size = %10d, ctime = %10d, ltime = %10d\n",FCB_address,file_name,FCB_read_start(fs,FCB_address), FCB_read_size(fs,FCB_address), FCB_read_ctime(fs, FCB_address), FCB_read_ltime(fs, FCB_address));
  }
  printf("===============PRINTING_FCB_BLOCK_INFO_END=================\n");
  // delete[] file_name;
}

/*
  Description:   For testing only, printing out all VCB info.
  Input:  N/A
  Output:   N/A
*/
__device__ void print_VCB(FileSystem *fs){
  printf("===============PRINTING_VCB_BLOCK_INFO=====================\n");
  for(int i = 0; i < 4096; i++){
    if(i % 8 == 0) printf("%4d ",i);
    for(int j = 0; j < 8; j++){
      uchar s = (fs->volume[i] >> (7-j)) & 0b00000001;
      if(s == 0) printf("x");
      else printf("|");
    }
    if(i % 8 == 7) printf("\n");
  }
  printf("===============PRINTING_VCB_BLOCK_INFO_END=================\n");
}


/*
  Description:   Check if there is continuous n free blocks. (unit: 32-bytes-large block)
  Input:  blocks
  Output: blocks (if memory compaction needed, then return -1)
*/
__device__ int VCB_Query(FileSystem *fs, u32 n){
  bool found = false;
  int current_cfree_block = 0;
  int total_free_block = 0;
  int result;
  for(int i = 0; i < fs->SUPERBLOCK_SIZE; i++){
    uchar unit = fs->volume[i];
    for(u32 j = 0; j < 8; j++){
      uchar bit = (unit >> (7-j)) & 0b00000001;
      if(bit == 0){
        current_cfree_block++;
        total_free_block++;
        if(current_cfree_block >= n){
          found = true;
          result = i*8 + j + 1 - current_cfree_block;
          break;
        }
      }
      else{
        current_cfree_block = 0;
      }
    }
    if(found) break;
  }
  if(!found){
    if(total_free_block < n) return -2;
    else return -1;
  }
  return result;
}

/*
  Description:   bit-operation on masking, supporting VCB_modification ONLY
  Input:  uchar # of VCB block
  Output: N/A
*/
__device__ void cover(FileSystem *fs, u32 layer, u32 start, u32 end, u32 value){
  uchar mask = 0;
  for(u32 j = start; j < end; j++){
    mask += (1 << (7 - j));
  }
  if(value) fs->volume[layer] |= mask; 
  else fs->volume[layer] &= ~mask; 
}

/*
  Description:   Set the [start, start+size] in VCB to be 0/1. 
  Input:  start (Unit:block), size (Unit: block)
  Output: N/A
*/
__device__ void VCB_modification(FileSystem *fs, u32 start, u32 size, u32 value){
  u32 start_i = start / 8;
  u32 start_j = start % 8;
  u32 end_i = (start + size) / 8;
  u32 end_j = (start + size) % 8;
  if(start_i == end_i){
    cover(fs, start_i, start_j, end_j, value);
  }
  else{
    cover(fs, start_i, start_j, 8, value);
    cover(fs, end_i, 0, end_j, value);
    for(u32 i = start_i+1; i < end_i; i++){
      fs->volume[i] = (value) ? 0xffffffff : 0x00000000;
    }
  }
}

/*
  Description:   Memory Compaction (a very time-comsuming job)
  Input:  N/A
  Output:   N/A
*/
__device__ void memory_compaction(FileSystem *fs){
  // TODO
  u32 FCB_address;
  u32 total_size = 0;
  u32 last_endpoint = 0;
  int block_size;
  u32 min_start, min_address;

  for(int i = 0; i < fs->VALID_BLOCK; i++){
    min_start = 99999;
    for(FCB_address = 0; FCB_address < fs->FCB_ENTRIES; FCB_address++){
      if(!FCB_read_validbit(fs,FCB_address)) continue;
      u32 start = FCB_read_start(fs, FCB_address);
      if(start < min_start && start >= last_endpoint){
        min_start = start;
        min_address = FCB_address;
      }
    }
    if(min_start > 40000) continue;
    block_size = (FCB_read_size(fs, min_address) -1) / fs->FCB_SIZE + 1;
    if(!block_size) continue;
    uchar *dest = &fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_ENTRIES * fs->FCB_SIZE + last_endpoint * fs->FCB_SIZE];
    uchar *source = &fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_ENTRIES * fs->FCB_SIZE + min_start * fs->FCB_SIZE];
    memcpy(dest, source, block_size*fs->FCB_SIZE);
    FCB_set_start(fs, min_address, last_endpoint);
    total_size += block_size;
    last_endpoint += block_size;
  }
  VCB_modification(fs, 0, total_size, 1);
  VCB_modification(fs, total_size, (fs->SUPERBLOCK_SIZE * 8 - total_size), 0);
}

__device__ u32 fs_open(FileSystem *fs, char *s, int op)
{
  // Search the FCB, see if there are 
  uchar file_name[20];
  bool found = false;
  int FCB_address;
  for(FCB_address = 0; FCB_address < fs->FCB_ENTRIES; FCB_address++){
    FCB_read_filename(fs, FCB_address, file_name);
    if(memcmp(file_name,(uchar*)s,20) == 0){
      if(FCB_read_validbit(fs, FCB_address)){
        // printf("Found\n");
        found = true;
        break;
      }
    }
  }
  if(!found){
    for(FCB_address = 0; FCB_address < fs->FCB_ENTRIES; FCB_address++){
      if(FCB_read_validbit(fs, FCB_address) == 0){
        found = true;
        break;
      }
    }
    FCB_set_filename(fs, FCB_address, (uchar*)s);
    FCB_set_validbit(fs, FCB_address, 1);
    FCB_set_start(fs, FCB_address, pow(2,16) - 1);   // No actual meaning, not involving memory compaction
    FCB_set_size(fs, FCB_address, 0);  
    FCB_set_ctime(fs, FCB_address);
    FCB_set_ltime(fs, FCB_address);
    fs->VALID_BLOCK++;
  }
  else{
    // Clean up the area.
    if(op == G_WRITE){
      u32 start = FCB_read_start(fs, FCB_address);
      int size = FCB_read_size(fs, FCB_address);
      VCB_modification(fs, start, (size - 1) / fs->FCB_SIZE + 1, 0);
    }
  }
  return (FCB_address + (op << 31));
}

__device__ void fs_read(FileSystem *fs, uchar *output, u32 size, u32 fp)
{
	/* Implement read operation here */
  u32 FCB_address = fp & 0x7fffffff;
  u32 op = (fp & 0x80000000) >> 31;
  assert(op == 0);
  u32 FCB_block_size = FCB_read_size(fs, FCB_address);
  if(FCB_block_size < size) printf("ERROR: FCB_address = %d, FCB_block_size < size, FCB_block_size = %d, size = %d\n",FCB_address,FCB_block_size,size);
  /* Read from storage */
  if(size == 0) return;
  u32 FCB_Start = FCB_read_start(fs, FCB_address);
  uchar *source = &fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_ENTRIES * fs->FCB_SIZE + FCB_Start * fs->FCB_SIZE];
  memcpy(output, source, size);
}

/*
  Description:   Implement write operation here
  Input:  input pointer, size, file descriptor
  Output:   N/A
*/
__device__ u32 fs_write(FileSystem *fs, uchar* input, u32 size, u32 fp)
{
  if(size == 0) return 0;
  u32 FCB_address = fp & 0x7fffffff;
  u32 op = (fp & 0x80000000) >> 31;
  assert(op == 1);
  u32 original_size = FCB_read_size(fs, FCB_address);
  int storage_address = FCB_read_start(fs, FCB_address);
  if(original_size < size){
    storage_address = VCB_Query(fs, (size - 1 ) / fs->FCB_SIZE + 1);
    if(storage_address <= -2) printf("Error! storage_address = %d\n",storage_address);
    if(storage_address == -1){
      memory_compaction(fs);
      storage_address = VCB_Query(fs, (size - 1 ) / fs->FCB_SIZE + 1);
    }
  }
  uchar *target = &fs->volume[fs->SUPERBLOCK_SIZE + fs->FCB_ENTRIES * fs->FCB_SIZE + storage_address * fs->FCB_SIZE];
  memcpy(target, input, size);
  FCB_set_start(fs, FCB_address, storage_address);
  FCB_set_size(fs, FCB_address, size);
  FCB_set_ltime(fs, FCB_address);
  VCB_modification(fs, storage_address, (size - 1) / fs->FCB_SIZE + 1, 1);
  return 0;
}

/*
  Description:    Implement LS_D and LS_S operation here
  Input:  LS_D | LS_S
  Output: N/A
*/
__device__ void fs_gsys(FileSystem *fs, int op)
{
  u32 FCB_address, ltime, size, current_max_address;
  u32 last_max_ltime = pow(2,24);
  u32 current_max_ltime = 0;
  u32 last_max_size = fs->MAX_FILE_SIZE + 1;
  u32 current_max_size = 0;
  uchar name[20];

  u32 ctime, current_min_address;
  u32 last_min_ctime = 0;
  u32 current_min_ctime = pow(2,24);

  bool found_inner;

  assert(op == LS_D || op == LS_S);
  if(op == LS_D){
    printf("===sort by modified time===\n");
    for(int i = 0; i < fs->VALID_BLOCK; i++){
      current_max_ltime = 0;
      for(FCB_address = 0; FCB_address < fs->FCB_ENTRIES; FCB_address++){
        if(!FCB_read_validbit(fs,FCB_address)) continue;
        ltime = FCB_read_ltime(fs, FCB_address);
        if(ltime < last_max_ltime && ltime >= current_max_ltime){
          current_max_ltime = ltime;
          current_max_address = FCB_address;
        }
      }
      FCB_read_filename(fs, current_max_address, name);
      current_max_size = FCB_read_size(fs, current_max_address);
      printf("%s\n",name);
      last_max_ltime = current_max_ltime;
    }
  }
  else{
    printf("===sort by file size===\n");
    for(int i = 0; i < fs->VALID_BLOCK; i++){
      current_max_size = 0;
      current_min_ctime = pow(2,24);
      found_inner = false;
      for(FCB_address = 0; FCB_address < fs->FCB_ENTRIES; FCB_address++){
        
        if(!FCB_read_validbit(fs,FCB_address)) continue;
        size = FCB_read_size(fs, FCB_address);
        ctime = FCB_read_ctime(fs, FCB_address);
        if(size < last_max_size && size > current_max_size){
          FCB_read_filename(fs,FCB_address, name);
          current_max_size = size;
          current_max_address = FCB_address;
          current_min_ctime = ctime;
        }
        else if(size == last_max_size){
          if(ctime > last_min_ctime){
            if(found_inner == false){
              current_min_ctime = pow(2,24);
              found_inner = true;
            } 
            if(ctime <= current_min_ctime){
              current_max_size = size;
              current_max_address = FCB_address;
              current_min_ctime = ctime;
            }
          }
        }
        else if(size == current_max_size){
          if(ctime < current_min_ctime){
            current_max_size = size;
            current_max_address = FCB_address;
            current_min_ctime = ctime;
          }
        }
      }
      FCB_read_filename(fs, current_max_address, name);
      printf("%s %d\n",name, current_max_size);
      last_min_ctime = current_min_ctime;
      last_max_size = current_max_size;
    }
  }
}

/*
  Description:   Implement rm operation here
  Input:  RM, Filename
  Output:   N/A
*/
__device__ void fs_gsys(FileSystem *fs, int op, char *s)
{
  assert(op == RM);
  uchar file_name[20];
  bool found = false;
  int FCB_address;
  for(FCB_address = 0; FCB_address < fs->FCB_ENTRIES; FCB_address++){
    FCB_read_filename(fs, FCB_address, file_name);
    if(memcmp(file_name,(uchar*)s,20) == 0){
      if(FCB_read_validbit(fs, FCB_address)){
        found = true;
        break;
      }
    }
  }
  if(found){
    FCB_set_validbit(fs, FCB_address, 0);
    fs->VALID_BLOCK--;
    if(!FCB_read_size(fs, FCB_address)) return;
    u32 start = FCB_read_start(fs, FCB_address);
    int block_size = (FCB_read_size(fs, FCB_address) - 1) / fs->FCB_SIZE + 1;
    VCB_modification(fs, start, block_size, 0);
  }
  else{
    printf("Error! The file '%s' does not exists.\n",s);
  }
}
